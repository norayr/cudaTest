
#include <hip/hip_runtime.h>


__global__ void cuda_Vec_Add(float* A, float* B, float* C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

void cudaVecAdd(int threads, float* a, float* b, float* c)
{
cuda_Vec_Add<<<1, threads>>>(a, b, c);
}
